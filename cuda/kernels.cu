
#include <hip/hip_runtime.h>
extern "C" __global__ void vector_add_kernel(float* result, const float* a, const float* b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] + b[idx];
    }
}

extern "C" __global__ void vector_multiply_kernel(float* result, const float* a, const float* b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] * b[idx];
    }
}

extern "C" __global__ void vector_sub_kernel(float* result, const float* a, const float* b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] - b[idx];
    }
}

extern "C" __global__ void vector_div_kernel(float* result, const float* a, const float* b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] / b[idx];
    }
}

// Add other kernels for exp, log, pow, sqrt, etc.